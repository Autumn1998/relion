#include "hip/hip_runtime.h"
#include "read_write_mrc.h"
#include "atom.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define FALSE 0
#define TRUE 1
using namespace std;

int PrjXYAngN;
int vol_pixel_num;
double iStart,iElaps;

int iLen = 256;

double cpuSecond(){
	struct timeval tp;
	gettimeofday(&tp,NULL);
	//sce + msec
	return (double)tp.tv_sec +(double )tp.tv_usec*1e-6;
}

__host__ int read_data(Volume *vol,Projection *prj,MrcHeader *in_head,MrcHeader *out_head,double *x_coef,double *y_coef,float *prj_real,float *vol_real,char *in_addr,char *angle_addr)
{
/***********************Malloc space**********************************************/
	//hipMallocManaged((void **)&vol,sizeof(Volume));
	//hipMallocManaged((void **)&prj,sizeof(Projection));
	//hipMallocManaged((void **)&in_head,sizeof(MrcHeader));
	//hipMallocManaged((void **)&out_head,sizeof(MrcHeader));
	vol = (Volume *)malloc(sizeof(Volume));
	prj = (Projection *)malloc(sizeof(Projection));
	in_head = (MrcHeader *)malloc(sizeof(MrcHeader));
	out_head = (MrcHeader *)malloc(sizeof(MrcHeader));

/*****************read head-file ande angle-file*************************************************************/
	FILE *in_file;
	in_file = fopen(in_addr,"r");
	if(!in_file){
		printf("Can not open in_file");
		return FALSE;	
	}
	mrc_read_head(in_file,in_head);
	fclose(in_file);
	printf("%d %d %d\n",in_head->nx,in_head->ny,in_head->nz);
	prj->X = in_head->nx;
	prj->Y = in_head->ny;
	prj->AngN = in_head->nz;
	
	vol->X=1467;
	vol->Y=1521;
	vol->Z=58;

	vol->Xstart=-259;
	vol->Xend=vol->Xstart+vol->X;
	vol->Ystart=-209;
	vol->Yend=vol->Ystart+vol->Y;
	vol->Zstart=-32;
	vol->Zend=vol->Zstart+vol->Z;

	mrc_init_head(out_head);
	out_head->nx=vol->X;
	out_head->ny=vol->Y;
	out_head->nz=vol->Z;

	out_head->nxstart=vol->Xstart;
	out_head->nystart=vol->Ystart;
	out_head->nzstart=vol->Zstart;

	out_head->mx=vol->X;
	out_head->my=vol->Y;
	out_head->mz=vol->Z;

	//hipMallocManaged((void **)&x_coef,sizeof(double)*prj->AngN*10);
	//hipMallocManaged((void **)&y_coef,sizeof(double)*prj->AngN*10);
	x_coef = (double *)malloc(sizeof(double)*prj->AngN*10);
	y_coef = (double *)malloc(sizeof(double)*prj->AngN*10);
	memset(x_coef, 0 , sizeof(double)*prj->AngN*10);
	memset(y_coef, 0 , sizeof(double)*prj->AngN*10);

	printf("%d\n",sizeof(double)*prj->AngN*10);

	FILE *angle_file;
	angle_file = fopen(angle_addr,"r");
	if(!angle_file){
		printf("Can not open angle_file");
		return FALSE;	
	}
	read_coef(x_coef, y_coef, angle_file);
	fclose(angle_file);
/***********************************************************************************************************/

/*****************************************read all data*****************************************************/
	PrjXYAngN = prj->X*prj->Y*prj->AngN;
	vol_pixel_num = vol->X*vol->Y*vol->Z;

	//hipMallocManaged((void **)&prj_real,sizeof(float)*PrjXYAngN);
	//hipMallocManaged((void **)&vol_real,sizeof(float)*vol_pixel_num);
	prj_real=(float *)malloc(sizeof(float)*PrjXYAngN);
	vol_real = (float *)malloc(sizeof(float)*vol_pixel_num);
	memset(prj_real, 0 , sizeof(float)*PrjXYAngN);
	memset(vol_real, 0 , sizeof(float)*vol_pixel_num);
	
	FILE *in_context=fopen(in_addr,"r");
	if(!in_context){
		printf("Can not open in_context");
		return FALSE;	
	}
	mrc_read_all(in_context,in_head,prj_real);
	fclose(in_context);
/*******************************************************************************************************/

	return TRUE;
}

/**
	Pixel pixel;
	Weight *comp_prj;
	comp_prj=(Weight *)malloc(sizeof(Weight));
	if((comp_prj=(Weight *)malloc(sizeof(Weight)))==NULL)
	{
		printf("Error with Function 'read_data()'!Can't malloc memery for 'comp_prj'!");
		return FALSE;
	}
        Slice_backproj_bilinear(pixel, prj, vol, comp_prj, x_coef, y_coef, prj_real,slc_data,Z_start,Z_end,slcN_per); //four-weight 
**/

/*
	int sizeofZ_per_block = vol->Z/process_num+1;
	int* Z_start = (int *)malloc(sizeof(int)*process_num);
	int* Z_end = (int *)malloc(sizeof(int)*process_num);//the start or end slice of reproject per process
	int* Z_per = (int *)malloc(sizeof(int)*process_num);
	printf("%d / %d = %d \n",vol->Z,process_num,sizeofZ_per_block);
	for(int i=0;i<process_num;i++)
	{
		Z_start[i] = vol->Zstart+i*sizeofZ_per_block;
		Z_end[i] = min(Z_start[i]+sizeofZ_per_block,vol->Zstart+vol->Z);
		Z_per[i] = Z_end[i]-Z_start[i];
		printf("For the %dth block, Z_start is %d, Z_end is %d,Z_per is %d\n",i,Z_start[i],Z_end[i],Z_per[i]);
	}
*/

__device__ void computeWeight(int x,int y,int z,int angle,double *x_coef,double *y_coef,Weight *weight)
{
	double res_x,res_y;
	int index = 4*angle;
	res_x = x_coef[index]+x_coef[index+1]*x+x_coef[index+2]*y+x_coef[index+3]*z;
	res_y = y_coef[index]+y_coef[index+1]*x+y_coef[index+2]*y+y_coef[index+3]*z;	

	weight->x_min = floor(res_x);
	weight->y_min = floor(res_y);

	weight->x_min_del = res_x - weight->x_min;
	weight->y_min_del = res_y - weight->y_min;
}

__global__ void backProjOnGPU(Projection *prj,Volume *vol,double *x_coef,double *y_coef,float *prj_real,float *vol_real,Weight *weight,int *test)
{
	double divisor;//分子
	double dividend;//分母
	int z = threadIdx.x+blockIdx.x*blockDim.x;
	if(z>=vol->Z) return;
	int x,y,index,angle,n;
	//test[z] = 1;
	//printf("Slice %d is beginning.\n",z);

	for(y=vol->Ystart;y<vol->Yend;y++)
	{
		for(x=vol->Xstart;x<vol->Xend;x++)
		{
		/*	divisor = 0;
			dividend = 0;
			index = (x-vol->Xstart)+(y-vol->Ystart)*vol->X;//+(z-vol->Zstart)*vol->X*vol->Y;
			//要计算的像素在三维结构中的坐标
			//if(x-vol->Xstart<2&&y-vol->Ystart<2&&index<vol->X)
			//printf("%d %d %d %d %d\n",index,x,vol->Xstart,y,vol->Ystart);
			for(angle=0;angle<prj->AngN;angle++)
			{
				computeWeight(x,y,z,angle,x_coef,y_coef,weight);
				//printf("%lf %lf \n",weight->x_min,weight->y_min);
				if(weight->x_min>=0 && weight->x_min<prj->X && weight->y_min>=0 && weight->y_min<prj->Y)//(x_min,y_min)
				{
					n = weight->x_min + weight->y_min*prj->X + angle*prj->X*prj->Y;
					divisor += (1-weight->x_min_del)*(1-weight->y_min_del)*prj_real[n];
					dividend += (1-weight->x_min_del)*(1-weight->y_min_del);
				}
				if(weight->x_min>=0 && weight->x_min<prj->X && weight->y_min>=0 && weight->y_min<prj->Y)//(x_min+1,y_min)
				{
					n = (weight->x_min+1) + weight->y_min*prj->X + angle*prj->X*prj->Y;
					divisor += weight->x_min_del*(1-weight->y_min_del)*prj_real[n];
					dividend += weight->x_min_del*(1-weight->y_min_del);
				}
				if(weight->x_min>=0 && weight->x_min<prj->X && weight->y_min>=0 && weight->y_min<prj->Y)//(x_min,y_min+1)
				{
					n = weight->x_min + (weight->y_min+1)*prj->X + angle*prj->X*prj->Y;
					divisor += (1-weight->x_min_del)*weight->y_min_del*prj_real[n];
					dividend += (1-weight->x_min_del)*weight->y_min_del;
				}
				if(weight->x_min>=0 && weight->x_min<prj->X && weight->y_min>=0 && weight->y_min<prj->Y)//(x_min+1,y_min+1)
				{
					n = (weight->x_min+1)+ (weight->y_min+1)*prj->X + angle*prj->X*prj->Y;
					divisor += weight->x_min_del*weight->y_min_del*prj_real[n];
					dividend += weight->x_min_del*weight->y_min_del;
				}
			}
			if(dividend!=0.0f)
			{
				vol_real[index] = (float)(divisor/dividend);
				//printf("vol_read[%d]:%f\n",index,vol_real[index]);
			}
			
			//if(19000<index&&index<19010)
			//if(index<10)
			//printf("%d-------------%lf\n",index,vol_real[index]);
		*/
		}
		
	}
	//test[z] = 2;
	//printf("Slice %d has been finished.\n",z);
}

void write_data(char *out_addr,MrcHeader *out_head,int Z_end,float *vol_real)
{
	FILE *out_file;
	out_file = fopen(out_addr,"w");
	if(!out_file){
		printf("Can not open in_file");
		return;	
	}
	mrc_write_head(out_file,out_head);
	printf("siezof out_head %ld \n",sizeof(out_head));
	mrc_write_all(out_file,out_head,Z_end,vol_real);
	mrc_update_head(out_file);
	fclose(out_file);
	return;
}

int main(int argc,char *argv[])
{
	iStart = cpuSecond();

	iLen = atoi(argv[1]);
	char* in_addr = argv[2];
	char* out_addr = argv[3];
	char* angle_addr = argv[4];
	//cout<<"arg1:"<<process_num<<" arg2:"<<in_addr<<"  arg3:"<<out_addr<<"  arg4:"<<angle_addr<<endl;
	
	/*******************Read data_in_host********************/
	Volume *h_vol;
	Projection *h_prj;
	MrcHeader *in_head,*out_head;
	double *h_x_coef;
	double *h_y_coef;
	float *h_prj_real,*h_vol_real;//pri_real is inputted data ande vol_real is calculated data

	read_data(h_vol,h_prj,in_head,out_head,h_x_coef,h_y_coef,h_prj_real,h_vol_real,in_addr,angle_addr);
	/******************************************************/

	/********************copy data from host to device************************/
	Volume *vol;
	Projection *prj;
	double *x_coef;
	double *y_coef;
	float *prj_real,*vol_real;
	Weight *weight;
	hipMalloc((void **)&vol,sizeof(Volume));
	hipMalloc((void **)&prj,sizeof(Projection));
	hipMalloc((void **)&x_coef,sizeof(double)*h_prj->AngN*10);
	hipMalloc((void **)&y_coef,sizeof(double)*h_prj->AngN*10);
	hipMalloc((void **)&prj_real,sizeof(float)*PrjXYAngN);
	hipMalloc((void **)&vol_real,sizeof(float)*vol_pixel_num);
	hipMalloc((void **)&weight,sizeof(Weight));
	
	hipMemcpy((void *)vol,(void *)h_vol,sizeof(Volume),hipMemcpyHostToDevice);
	hipMemcpy((void *)prj,(void *)h_prj,sizeof(Projection),hipMemcpyHostToDevice);
	hipMemcpy((void *)x_coef,(void *)h_x_coef,sizeof(double)*h_prj->AngN*10,hipMemcpyHostToDevice);
	hipMemcpy((void *)y_coef,(void *)h_y_coef,sizeof(double)*h_prj->AngN*10,hipMemcpyHostToDevice);
	hipMemcpy((void *)prj_real,(void *)h_prj_real,sizeof(float)*PrjXYAngN,hipMemcpyHostToDevice);
	/********************************************************************/
	
	/*********************Debug**********************/
	printf("%d\n",h_vol->Z);
	int *test = (int *)malloc(sizeof(int)*h_vol->Z);
	//hipMallocManaged((void **)&test,vol->Z*sizeof(int)*10);
	for(int i=0;i<h_vol->Z;i++) test[i] = 0;
	int *d_test;
	hipMalloc((void **)&d_test,sizeof(int)*h_vol->Z);
	hipMemcpy(d_test,test,sizeof(int)*h_vol->Z,hipMemcpyHostToDevice);
	/************************************************/


	/************run on GPU*********************/
	dim3 block(iLen);
	dim3 grid((h_vol->Z+block.x-1)/block.x);
	backProjOnGPU<<<grid,block>>>(prj,vol,x_coef,y_coef,prj_real,vol_real,weight,test);	
	//hipDeviceSynchronize();
	/******************************************/

	/************copy result from Device to host***********/
	hipMemcpy((void *)h_vol_real,(void *)vol_real,sizeof(float)*vol_pixel_num,hipMemcpyDeviceToHost);	
	/*******/

	//write_data(out_addr,out_head,h_vol->Z,h_vol_real);

	/************Debug******************/
	hipMemcpy((void *)test,(void *)d_test,sizeof(int)*h_vol->Z,hipMemcpyDeviceToHost);
	for(int i=0;i<vol->Z;i++) printf("%d ",test[i]);
	/******************************/

	/*************free space****************************/
	hipFree(vol);
	hipFree(prj);
	hipFree(x_coef);
	hipFree(y_coef);
	hipFree(prj_real);
	hipFree(vol_real);
	free(h_vol);
	free(h_prj);
	free(h_x_coef);
	free(h_y_coef);
	free(prj_real);
	free(vol_real);
	free(in_head);
	free(out_head);
	/****************************************************/

	iElaps = cpuSecond()-iStart;
	printf("Host time elapsed:%lfsec\n",iElaps);
	return 0;
}

