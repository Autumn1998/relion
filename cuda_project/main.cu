#include "hip/hip_runtime.h"
#include "read_write_mrc.h"
#include "atom.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define FALSE 0
#define TRUE 1
using namespace std;

int PrjXYAngN;
int vol_pixel_num;
double iStart,iElaps;

int iLen = 256;

double cpuSecond(){
	struct timeval tp;
	gettimeofday(&tp,NULL);
	//sce + msec
	return (double)tp.tv_sec +(double )tp.tv_usec*1e-6;
}

int read_head_data(Volume *vol,Projection *prj,MrcHeader *in_head,MrcHeader *out_head,char *in_addr)
{
/*****************read head-file ande angle-file*************************************************************/	
	FILE *in_file;
	in_file = fopen(in_addr,"r");
	if(!in_file){
		printf("Can not open in_file");
		return FALSE;	
	}
	mrc_read_head(in_file,in_head);
	fclose(in_file);
	printf("%d %d %d\n",in_head->nx,in_head->ny,in_head->nz);
	prj->X = in_head->nx;
	prj->Y = in_head->ny;
	prj->AngN = in_head->nz;
	
	vol->X=1467;
	vol->Y=1521;
	vol->Z=58;

	vol->Xstart=-259;
	vol->Xend=vol->Xstart+vol->X;
	vol->Ystart=-209;
	vol->Yend=vol->Ystart+vol->Y;
	vol->Zstart=-32;
	vol->Zend=vol->Zstart+vol->Z;

	mrc_init_head(out_head);

	out_head->nx=vol->X;
	out_head->ny=vol->Y;
	out_head->nz=vol->Z;

	out_head->nxstart=vol->Xstart;
	out_head->nystart=vol->Ystart;
	out_head->nzstart=vol->Zstart;

	out_head->mx=vol->X;
	out_head->my=vol->Y;
	out_head->mz=vol->Z;

	printf("%d 0\n",vol->Z);
	return TRUE;
}


int read_txbr_data(double *x_coef,double *y_coef,char *angle_addr)
{
	FILE *angle_file;
	angle_file = fopen(angle_addr,"r");
	if(!angle_file){
		printf("Can not open angle_file");
		return FALSE;	
	}
	read_coef(x_coef, y_coef, angle_file);
	fclose(angle_file);
}

int read_all_data(MrcHeader *in_head,float *prj_real,char *in_addr)
{	
	FILE *in_context=fopen(in_addr,"r");
	if(!in_context){
		printf("Can not open in_context");
		return FALSE;	
	}
	mrc_read_all(in_context,in_head,prj_real);
	fclose(in_context);
}

/*
	int sizeofZ_per_block = vol->Z/process_num+1;
	int* Z_start = (int *)malloc(sizeof(int)*process_num);
	int* Z_end = (int *)malloc(sizeof(int)*process_num);//the start or end slice of reproject per process
	int* Z_per = (int *)malloc(sizeof(int)*process_num);
	printf("%d / %d = %d \n",vol->Z,process_num,sizeofZ_per_block);
	for(int i=0;i<process_num;i++)
	{
		Z_start[i] = vol->Zstart+i*sizeofZ_per_block;
		Z_end[i] = min(Z_start[i]+sizeofZ_per_block,vol->Zstart+vol->Z);
		Z_per[i] = Z_end[i]-Z_start[i];
		printf("For the %dth block, Z_start is %d, Z_end is %d,Z_per is %d\n",i,Z_start[i],Z_end[i],Z_per[i]);
	}
*/


__global__ void backProjOnGPU(Projection *prj,Volume *vol,double *x_coef,double *y_coef,float *prj_real,float *vol_real,int *test)
{
	double divisor;//分子
	double dividend;//分母
	int z = threadIdx.x+blockIdx.x*blockDim.x + vol->Zstart;
	if(z!=vol->Zstart) return;
	if(z>=vol->Zend) return;
	int x,y,index,angle,n;	
	test[z] = 1;
	for(y=vol->Ystart;y<vol->Ystart+vol->Y;y++)
	{
		for(x=vol->Xstart;x<0;x++)
		{
			divisor = 0;
			dividend = 0;
			for(angle=0;angle<prj->AngN;angle++)
			{
				double res_x,res_y,x_min_del,y_min_del;
				int id = 4*angle,x_min,y_min;
				res_x = x_coef[id]+x_coef[id+1]*x+x_coef[id+2]*y+x_coef[id+3]*z;
				res_y = y_coef[id]+y_coef[id+1]*x+y_coef[id+2]*y+y_coef[id+3]*z;	
				x_min = floor(res_x);
				y_min = floor(res_y);
				x_min_del = res_x - x_min;
				y_min_del = res_y - y_min;
				
				if(x_min>=0 && x_min<prj->X && y_min>=0 && y_min<prj->Y)//(x_min,y_min)
				{
					n = x_min + y_min*prj->X + angle*prj->X*prj->Y;
					divisor += (1-x_min_del)*(1-y_min_del)*prj_real[n];
					dividend += (1-x_min_del)*(1-y_min_del);
				}
				if(x_min>=0 && x_min<prj->X && y_min>=0 && y_min<prj->Y)//(x_min+1,y_min)
				{
					n = (x_min+1) + y_min*prj->X + angle*prj->X*prj->Y;
					divisor += x_min_del*(1-y_min_del)*prj_real[n];
					dividend += x_min_del*(1-y_min_del);
				}
				if(x_min>=0 && x_min<prj->X && y_min>=0 && y_min<prj->Y)//(x_min,y_min+1)
				{
					n = x_min + (y_min+1)*prj->X + angle*prj->X*prj->Y;
					divisor += (1-x_min_del)*y_min_del*prj_real[n];
					dividend += (1-x_min_del)*y_min_del;
				}
				if(x_min>=0 && x_min<prj->X && y_min>=0 && y_min<prj->Y)//(x_min+1,y_min+1)
				{
					n = (x_min+1)+ (y_min+1)*prj->X + angle*prj->X*prj->Y;
					divisor += x_min_del*y_min_del*prj_real[n];
					dividend += x_min_del*y_min_del;
				}
			}
			index = (x-vol->Xstart)+(y-vol->Ystart)*vol->X+(z-vol->Zstart)*vol->X*vol->Y;
			printf("%d\n",index);
			if(dividend!=0.0f)
			{
				vol_real[index] = (float)(divisor/dividend);
				//printf("vol_read[%d]:%f\n",index,vol_real[index]);
			}
		}
		
	}
	test[z] = 2;	
}

void write_data(char *out_addr,MrcHeader *out_head,int Z_end,float *vol_real)
{
	FILE *out_file;
	out_file = fopen(out_addr,"w");
	if(!out_file){
		printf("Can not open in_file");
		return;	
	}
	mrc_write_head(out_file,out_head);
	printf("siezof out_head %ld \n",sizeof(out_head));
	mrc_write_all(out_file,out_head,Z_end,vol_real);
	mrc_update_head(out_file);
	fclose(out_file);
	return;
}

__global__ void testOnGPU(int *test)
{
	int z = threadIdx.x+blockIdx.x*blockDim.x;
	if(z>=58) return;
	test[z] = 1;
}

int main(int argc,char *argv[])
{
	iStart = cpuSecond();

	iLen = atoi(argv[1]);
	char* in_addr = argv[2];
	char* out_addr = argv[3];
	char* angle_addr = argv[4];
	//cout<<"arg1:"<<process_num<<" arg2:"<<in_addr<<"  arg3:"<<out_addr<<"  arg4:"<<angle_addr<<endl;
	
	Volume *vol;
	Projection *prj;
	MrcHeader *in_head,*out_head;
	double *x_coef;
	double *y_coef;
	float *prj_real,*vol_real;//pri_real is inputted data ande vol_real is calculated data
	int *test;

/*************Head file read ande malloc space*******************/
	hipMallocManaged((void **)&vol,sizeof(Volume));
	hipMallocManaged((void **)&prj,sizeof(Projection));
	hipMallocManaged((void **)&in_head,sizeof(MrcHeader));
	hipMallocManaged((void **)&out_head,sizeof(MrcHeader));

	read_head_data(vol,prj,in_head,out_head,in_addr);

/********************************************************************/


/*************TXBR file read ande malloc space*******************/
	hipMallocManaged((void **)&test,sizeof(int)*vol->Z);
	memset(test,0,sizeof(int)*vol->Z);
	hipMallocManaged((void **)&x_coef,sizeof(double)*prj->AngN*10);
	memset(x_coef, 0 , sizeof(double)*prj->AngN*10);
	//printf("%d",sizeof(double)*prj->AngN*10);
	hipMallocManaged((void **)&y_coef,sizeof(double)*prj->AngN*10);
	memset(y_coef, 0 , sizeof(double)*prj->AngN*10);

	read_txbr_data(x_coef,y_coef,angle_addr);

/********************************************************************/


/*************Reminding data read ande malloc space*******************/

	PrjXYAngN = prj->X*prj->Y*prj->AngN;
	vol_pixel_num = vol->X*vol->Y*vol->Z;
	/*for input file*/
	hipMallocManaged((void **)&prj_real,sizeof(float)*PrjXYAngN);
	memset(prj_real, 0 , sizeof(float)*PrjXYAngN);
	/*for output file*/
	hipMallocManaged((void **)&vol_real,sizeof(float)*vol_pixel_num);
	memset(vol_real, 0 , sizeof(float)*vol_pixel_num);	
	read_all_data(in_head,prj_real, in_addr);
/********************************************************************/

	printf("%d 1\n",vol->Z);
	//for(int i=0;i<vol->Z;i++ ) printf("%d",test[i]);
	dim3 block(iLen);
	dim3 grid((vol->Z+block.x-1)/block.x);
	//hipDeviceSynchronize();
	backProjOnGPU<<<grid,block>>>(prj,vol,x_coef,y_coef,prj_real,vol_real,test);	
	//testOnGPU<<<grid,block>>>(test);
	hipDeviceSynchronize();
	
	//write_data(out_addr,out_head,vol->Zend-vol->Zstart,vol_real);
	
	printf("%d 2\n",vol->Z);
	for(int qwe = 0;qwe<vol->Z;qwe++) printf("%d",test[qwe]);
	//hipDeviceReset();//重置CUDA设备释放程序占用的资源

	iElaps = cpuSecond()-iStart;
	printf("Host time elapsed:%lfsec\n",iElaps);
	return 0;
}

