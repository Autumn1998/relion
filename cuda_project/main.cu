#include "hip/hip_runtime.h"
#include "read_write_mrc.h"
#include "atom.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define FALSE 0
#define TRUE 1
#define checkCudaErrors( a ) do { \
	if (hipSuccess != (a)) { \
	fprintf(stderr, "Cuda runtime error in line %d of file %s \
	: %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
	exit(EXIT_FAILURE); \
	} \
	} while(0);
using namespace std;

int PrjXYAngN;
int vol_pixel_num;
double iStart,iElaps;

int iLen = 32;
int SIRT_ITER_NUM = 4;
float ITER_STEP_LENGTH = 0.2;

double cpuSecond(){
	struct timeval tp;
	gettimeofday(&tp,NULL);
	//sce + msec
	return (double)tp.tv_sec +(double )tp.tv_usec*1e-6;
}

int read_head_data(Volume *vol,Projection *prj,MrcHeader *in_head,MrcHeader *out_head,char *in_addr)
{
/*****************read head-file ande angle-file*************************************************************/	
	FILE *in_file;
	in_file = fopen(in_addr,"r");
	if(!in_file){
		printf("Can not open in_file");
		return FALSE;	
	}
	mrc_read_head(in_file,in_head);
	fclose(in_file);
	printf("%d %d %d\n",in_head->nx,in_head->ny,in_head->nz);
	prj->X = in_head->nx;
	prj->Y = in_head->ny;
	prj->AngN = in_head->nz;
	
	vol->X=1467;
	vol->Y=1521;
	vol->Z=58;

	vol->Xstart=-259;
	vol->Xend=vol->Xstart+vol->X;
	vol->Ystart=-209;
	vol->Yend=vol->Ystart+vol->Y;
	vol->Zstart=-32;
	vol->Zend=vol->Zstart+vol->Z;

	mrc_init_head(out_head);

	out_head->nx=vol->X;
	out_head->ny=vol->Y;
	out_head->nz=vol->Z;

	out_head->nxstart=vol->Xstart;
	out_head->nystart=vol->Ystart;
	out_head->nzstart=vol->Zstart;

	out_head->mx=vol->X;
	out_head->my=vol->Y;
	out_head->mz=vol->Z;

	printf("%d %d %d 0\n",out_head->nx,out_head->ny,out_head->nz);
	return TRUE;
}


int read_txbr_data(double *x_coef,double *y_coef,char *angle_addr)
{
	FILE *angle_file;
	angle_file = fopen(angle_addr,"r");
	if(!angle_file){
		printf("Can not open angle_file");
		return FALSE;	
	}
	read_coef(x_coef, y_coef, angle_file);
	fclose(angle_file);
	return true;
}

int read_all_data(MrcHeader *in_head,float *prj_real,char *in_addr)
{	
	FILE *in_context=fopen(in_addr,"r");
	if(!in_context){
		printf("Can not open in_context");
		return FALSE;	
	}
	mrc_read_all(in_context,in_head,prj_real);
	fclose(in_context);
	return true;
}

/*
	int sizeofZ_per_block = vol->Z/process_num+1;
	int* Z_start = (int *)malloc(sizeof(int)*process_num);
	int* Z_end = (int *)malloc(sizeof(int)*process_num);//the start or end slice of reproject per process
	int* Z_per = (int *)malloc(sizeof(int)*process_num);
	printf("%d / %d = %d \n",vol->Z,process_num,sizeofZ_per_block);
	for(int i=0;i<process_num;i++)
	{
		Z_start[i] = vol->Zstart+i*sizeofZ_per_block;
		Z_end[i] = min(Z_start[i]+sizeofZ_per_block,vol->Zstart+vol->Z);
		Z_per[i] = Z_end[i]-Z_start[i];
		printf("For the %dth block, Z_start is %d, Z_end is %d,Z_per is %d\n",i,Z_start[i],Z_end[i],Z_per[i]);
	}
*/


__global__ void backProjOnGPU(Projection *prj,Volume *vol,double *x_coef,double *y_coef,float *prj_real,float *vol_real,float iter_step_length)
{
	double divisor;//分子
	double dividend;//分母
	int x = threadIdx.x+blockIdx.x*blockDim.x +vol->Xstart;
	int y = threadIdx.y+blockIdx.y*blockDim.y +vol->Ystart;
	//printf("%d %d\n ",y,z);
	if(x>=vol->Xend || y>=vol->Yend) return;
	int z,index,angle,n;	
	for(z=vol->Zstart;z<vol->Zend;z++)
	{
		divisor = 0;
		dividend = 0;
		for(angle=0;angle<prj->AngN;angle++)
		{
			double res_x,res_y,x_min_del,y_min_del;
			int id = 4*angle,x_min,y_min;
			res_x = x_coef[id]+x_coef[id+1]*x+x_coef[id+2]*y+x_coef[id+3]*z;
			res_y = y_coef[id]+y_coef[id+1]*x+y_coef[id+2]*y+y_coef[id+3]*z;	
			x_min = floor(res_x);
			y_min = floor(res_y);
			x_min_del = res_x - x_min;
			y_min_del = res_y - y_min;
			
			if(x_min>=0 && x_min<prj->X && y_min>=0 && y_min<prj->Y)//(x_min,y_min)
			{
				n = x_min + y_min*prj->X + angle*prj->X*prj->Y;
				divisor += (1-x_min_del)*(1-y_min_del)*prj_real[n];
				dividend += (1-x_min_del)*(1-y_min_del);
			}
			if(x_min+1>=0 && x_min+1<prj->X && y_min>=0 && y_min<prj->Y)//(x_min+1,y_min)
			{
				n = (x_min+1) + y_min*prj->X + angle*prj->X*prj->Y;
				divisor += x_min_del*(1-y_min_del)*prj_real[n];
				dividend += x_min_del*(1-y_min_del);
			}
			if(x_min>=0 && x_min<prj->X && y_min+1>=0 && y_min+1<prj->Y)//(x_min,y_min+1)
			{
				n = x_min + (y_min+1)*prj->X + angle*prj->X*prj->Y;
				divisor += (1-x_min_del)*y_min_del*prj_real[n];
				dividend += (1-x_min_del)*y_min_del;
			}
			if(x_min+1>=0 && x_min+1<prj->X && y_min+1>=0 && y_min+1<prj->Y)//(x_min+1,y_min+1)
			{
				n = (x_min+1)+ (y_min+1)*prj->X + angle*prj->X*prj->Y;
				divisor += x_min_del*y_min_del*prj_real[n];
				dividend += x_min_del*y_min_del;
			}
		}
		if(dividend!=0.0f)
		{
			index = (x-vol->Xstart)+(y-vol->Ystart)*vol->X+(z-vol->Zstart)*vol->X*vol->Y;
			atomicAdd(&vol_real[index], (float)(divisor/dividend)*iter_step_length);
			//vol_real[index] = (float)(divisor/dividend);
			//if(index>vol->X*vol->Y+90000&&index<vol->X*vol->Y+90500)
			//	printf("vol_real[%d]:%f\n",index,vol_real[index]);
		}
	}	
}

__global__ void reProjOnGPU(Projection *prj,Volume *vol,double *x_coef,double *y_coef,float *vol_real,float *iter_prj_divisor,float *iter_prj_dividend)
{
	
	int x = threadIdx.x+blockIdx.x*blockDim.x +vol->Xstart;
	int y = threadIdx.y+blockIdx.y*blockDim.y +vol->Ystart;
	if(x>=vol->Xend || y>=vol->Yend) return;
	int z,index,angle,n;	
	for(z=vol->Zstart;z<vol->Zend;z++)
	{
		index = (x-vol->Xstart)+(y-vol->Ystart)*vol->X+(z-vol->Zstart)*vol->X*vol->Y;
		for(angle=0;angle<prj->AngN;angle++)
		{
			double res_x,res_y,x_min_del,y_min_del;
			int id = 4*angle,x_min,y_min;
			res_x = x_coef[id]+x_coef[id+1]*x+x_coef[id+2]*y+x_coef[id+3]*z;
			res_y = y_coef[id]+y_coef[id+1]*x+y_coef[id+2]*y+y_coef[id+3]*z;	
			x_min = floor(res_x);
			y_min = floor(res_y);
			x_min_del = res_x - x_min;
			y_min_del = res_y - y_min;
			
			if(x_min>=0 && x_min<prj->X && y_min>=0 && y_min<prj->Y)//(x_min,y_min)
			{
				n = x_min + y_min*prj->X + angle*prj->X*prj->Y;
				atomicAdd(&iter_prj_divisor[n], (1-x_min_del)*(1-y_min_del)*vol_real[index]);
				atomicAdd(&iter_prj_dividend[n], (1-x_min_del)*(1-y_min_del));
			}
			if(x_min+1>=0 && x_min+1<prj->X && y_min>=0 && y_min<prj->Y)//(x_min+1,y_min)
			{
				n = (x_min+1) + y_min*prj->X + angle*prj->X*prj->Y;
				atomicAdd(&iter_prj_divisor[n], x_min_del*(1-y_min_del)*vol_real[index]);
				atomicAdd(&iter_prj_dividend[n], x_min_del*(1-y_min_del));
			}
			if(x_min>=0 && x_min<prj->X && y_min+1>=0 && y_min+1<prj->Y)//(x_min,y_min+1)
			{
				n = x_min + (y_min+1)*prj->X + angle*prj->X*prj->Y;
				atomicAdd(&iter_prj_divisor[n], (1-x_min_del)*y_min_del*vol_real[index]);
				atomicAdd(&iter_prj_dividend[n], (1-x_min_del)*y_min_del);
			}
			if(x_min+1>=0 && x_min+1<prj->X && y_min+1>=0 && y_min+1<prj->Y)//(x_min+1,y_min+1)
			{
				n = (x_min+1)+ (y_min+1)*prj->X + angle*prj->X*prj->Y;
				atomicAdd(&iter_prj_divisor[n], x_min_del*y_min_del*vol_real[index]);
				atomicAdd(&iter_prj_dividend[n], x_min_del*y_min_del);
			}
		}
	}
}

int update_head(float *vol_real,MrcHeader *head)
{
	long double sum=0,amin,amax,amean;
	int prj_size=head->nx*head->ny,i,j;
	printf("updating head(FLOAT)...\n");
	amax = amin = vol_real[0];
	for(j = 0;j<head->nz;j++)
	{
		amean = 0;
		//printf("%d :%f\n",j,vol_real[90499]);
		for(i = 0;i<prj_size;i++)
		{
			int tmp_index = i+j*prj_size;
			if(vol_real[tmp_index]>amax) amax = vol_real[tmp_index];
			if(vol_real[tmp_index]<amin) amin = vol_real[tmp_index];
			amean+=vol_real[tmp_index];
		}
		amean/=prj_size;
		sum += amean;
	}
	amean = sum/head->nz;
	head->amin=amin;
	head->amax=amax;
	head->amean=amean;
	printf("head->amin is %f, head->amax is %f, head->amean is %f\n",head->amin, head->amax, head->amean);
	return true;
}

__global__ void computePrjError(Projection *prj,float *prj_real,float *iter_prj_divisor,float *iter_prj_dividend)
{
	int y = threadIdx.x+blockIdx.x*blockDim.x;
	int z = threadIdx.y+blockIdx.y*blockDim.y;
	if(y>=prj->Y || z>=prj->AngN) return;
	int x,index;	
	for(x=0;x<prj->X;x++)
	{
		index = x+y*prj->X+z*prj->X*prj->Y;
		if(iter_prj_dividend[index]!=0)
			iter_prj_divisor[index] /= iter_prj_dividend[index];
		iter_prj_divisor[index] = prj_real[index]-iter_prj_divisor[index];
	}
}

void write_data(char *out_addr,MrcHeader *out_head,float *vol_real)
{
	clean_file(out_addr);
	FILE *out_file;
	out_file = fopen(out_addr,"r+");
	if(!out_file){
		printf("Can not open out_file!\n");
		return;	
	}
	mrc_write_head(out_file,out_head);

	//printf("siezof out_head %ld \n",sizeof(MrcHeader));
	mrc_write_all(out_file,out_head,vol_real);
	printf("%d %d %d 1\n",out_head->nx,out_head->ny,out_head->nz);
	
	//mrc_update_head(out_file);
	fclose(out_file);
	return;
}

int main(int argc,char *argv[])
{
	iStart = cpuSecond();

	iLen = atoi(argv[1]);
	SIRT_ITER_NUM = atoi(argv[2]);
	ITER_STEP_LENGTH = atof(argv[3]);
	char* in_addr = argv[4];
	char* out_addr = argv[5];
	char* angle_addr = argv[6];
	//cout<<"arg1:"<<process_num<<" arg2:"<<in_addr<<"  arg3:"<<out_addr<<"  arg4:"<<angle_addr<<endl;
	
	Volume *vol;
	Projection *prj;
	MrcHeader *in_head,*out_head;
	double *x_coef;
	double *y_coef;
	float *prj_real,*vol_real;//pri_real is inputted data ande vol_real is calculated data

/*************Head file read ande malloc space*******************/
	hipMallocManaged((void **)&vol,sizeof(Volume));
	hipMallocManaged((void **)&prj,sizeof(Projection));
	hipMallocManaged((void **)&in_head,sizeof(MrcHeader));
	hipMallocManaged((void **)&out_head,sizeof(MrcHeader));

	read_head_data(vol,prj,in_head,out_head,in_addr);
/********************************************************************/


/*************TXBR file read ande malloc space*******************/
	hipMallocManaged((void **)&x_coef,sizeof(double)*prj->AngN*10);
	memset(x_coef, 0 , sizeof(double)*prj->AngN*10);
	//printf("%d",sizeof(double)*prj->AngN*10);
	hipMallocManaged((void **)&y_coef,sizeof(double)*prj->AngN*10);
	memset(y_coef, 0 , sizeof(double)*prj->AngN*10);

	read_txbr_data(x_coef,y_coef,angle_addr);
/********************************************************************/


/*************Reminding data read ande malloc space*******************/
	PrjXYAngN = prj->X*prj->Y*prj->AngN;
	vol_pixel_num = vol->X*vol->Y*vol->Z;
	printf("vol_pixel_num:%d\n",vol_pixel_num);
	printf("PriXYAngN:%d\n",PrjXYAngN);
	/*for input file*/
	hipMallocManaged((void **)&prj_real,sizeof(float)*PrjXYAngN);
	memset(prj_real, 0 , sizeof(float)*PrjXYAngN);
	/*for output file*/
	hipMallocManaged((void **)&vol_real,sizeof(float)*vol_pixel_num);
	memset(vol_real, 0 , sizeof(float)*vol_pixel_num);	
	read_all_data(in_head,prj_real, in_addr);
/********************************************************************/

/*****************back projection (initial modle)*****************************/
	printf("%d 1\n",iLen);
	dim3 block(iLen,iLen);
	dim3 grid((vol->X+block.x-1)/block.x,(vol->Y+block.y-1)/block.y);
	dim3 grid_prj((prj->X+block.x-1)/block.x,(prj->Y+block.y-1)/block.y);
	//hipDeviceSynchronize();
	backProjOnGPU<<<grid,block>>>(prj,vol,x_coef,y_coef,prj_real,vol_real,1);	
	hipDeviceSynchronize();
/*****************************************************************************/

/*********************DATA space needed by SIRT***********************/
	float *iter_prj_divisor,*iter_prj_dividend;
	checkCudaErrors(hipMallocManaged((void **)&iter_prj_divisor,sizeof(float)*PrjXYAngN));
	checkCudaErrors(hipMallocManaged((void **)&iter_prj_dividend,sizeof(float)*PrjXYAngN));
/*********************************************************************/

/*******************SIRT************************/
	for(int i=0;i<SIRT_ITER_NUM;i++)
	{
		memset(iter_prj_divisor,0,sizeof(float)*PrjXYAngN);
		memset(iter_prj_dividend,0,sizeof(float)*PrjXYAngN);
		reProjOnGPU<<<grid,block>>>(prj,vol,x_coef,y_coef,vol_real,iter_prj_divisor,iter_prj_dividend);
		hipDeviceSynchronize();
	/*	for(int index=0;index<PrjXYAngN;index++)
		{
			if(iter_prj_dividend[index]!=0)
				iter_prj_divisor[index] /= iter_prj_dividend[index];
			iter_prj_divisor[index] = prj_real[index]-iter_prj_divisor[index];
		}
	 */
		computePrjError<<<grid_prj,block>>>(prj,prj_real,iter_prj_divisor,iter_prj_dividend);
		hipDeviceSynchronize();
		backProjOnGPU<<<grid,block>>>(prj,vol,x_coef,y_coef,iter_prj_divisor,vol_real,ITER_STEP_LENGTH);
		hipDeviceSynchronize();
		printf("Iteration %d finished..\n",i);
	}
/**********************************************/

	update_head(vol_real,out_head);
	write_data(out_addr,out_head,vol_real);
	
	hipDeviceReset();//重置CUDA设备释放程序占用的资源

	iElaps = cpuSecond()-iStart;
	printf("Host time elapsed:%lfsec\n",iElaps);
	return 0;
}

